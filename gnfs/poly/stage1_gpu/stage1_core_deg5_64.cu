#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "stage1_core_deg5_64.h"

#ifdef __cplusplus
extern "C" {
#endif

#if 0
#define MONTMUL_RADIX 24
#else
#define MONTMUL_RADIX 32
#endif

#define MIN(x, y) ((x) < (y) ? (x) : (y))

/*------------------------------------------------------------------------*/
__device__ uint64 
modsub(uint64 a, uint64 b, uint64 p) 
{
	uint64 t = 0, tr;
	tr = a - b;
	if (tr > a)
		t = p;
	return tr + t;
}

/*------------------------------------------------------------------------*/
__device__ uint32 
modinv(uint32 a, uint32 p) {

	uint32 ps1, ps2, dividend, divisor, rem, q, t;
	uint32 parity;

	q = 1; rem = a; dividend = p; divisor = a;
	ps1 = 1; ps2 = 0; parity = 0;

	while (divisor > 1) {
		rem = dividend - divisor;
		t = rem - divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t;
		if (rem >= divisor) {
			q = dividend / divisor;
			rem = dividend - q * divisor;
			q *= ps1;
		} } } } } } } } }

		q += ps2;
		parity = ~parity;
		dividend = divisor;
		divisor = rem;
		ps2 = ps1;
		ps1 = q;
	}
	
	if (parity == 0)
		return ps1;
	else
		return p - ps1;
}

/*------------------------------------------------------------------------*/
__device__ uint32 
montmul_w(uint32 n) {

#if MONTMUL_RADIX == 24
	uint32 res = 8 - (n % 8);
	res = __umul24(res, 2 + __umul24(n, res));
	res = __umul24(res, 2 + __umul24(n, res));
	return __umul24(res, 2 + __umul24(n, res));
#else /* MONTMUL_RADIX == 32 */
	uint32 res = 2 + n;
	res = res * (2 + n * res);
	res = res * (2 + n * res);
	res = res * (2 + n * res);
	return res * (2 + n * res);
#endif
}

/*------------------------------------------------------------------------*/
#define LOW24 0xffffff

__device__ uint64 
montmul(uint64 a, uint64 b,
		uint64 n, uint32 w) {

#if MONTMUL_RADIX == 24
	uint32 a0 = (uint32)a & LOW24;
	uint32 a1 = (uint32)(a >> 24) & LOW24;
	uint32 a2 = (uint32)(a >> 48);
	uint32 b0 = (uint32)b & LOW24;
	uint32 b1 = (uint32)(b >> 24) & LOW24;
	uint32 b2 = (uint32)(b >> 48);
	uint32 n0 = (uint32)n & LOW24;
	uint32 n1 = (uint32)(n >> 24) & LOW24;
	uint32 n2 = (uint32)(n >> 48);
	uint32 q0, q1, q2;
	uint64 acc;
	uint32 prod_lo, prod_hi;
	uint64 r;

	prod_lo = __umul24(a0, b0);
	prod_hi = __umulhi(a0, b0);
	acc = (uint64)prod_hi << 32 | prod_lo;
	q0 = __umul24((uint32)acc, w) & LOW24;

	prod_lo = __umul24(q0, n0);
	prod_hi = __umulhi(q0, n0);
	acc = (acc + ((uint64)prod_hi << 32 | prod_lo)) >> 24;

	prod_lo = __umul24(a0, b1);
	prod_hi = __umulhi(a0, b1);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(a1, b0);
	prod_hi = __umulhi(a1, b0);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(q0, n1);
	prod_hi = __umulhi(q0, n1);
	acc += (uint64)prod_hi << 32 | prod_lo;
	q1 = __umul24((uint32)acc, w) & LOW24;

	prod_lo = __umul24(q1, n0);
	prod_hi = __umulhi(q1, n0);
	acc = (acc + ((uint64)prod_hi << 32 | prod_lo)) >> 24;

	prod_lo = __umul24(a0, b2);
	prod_hi = __umulhi(a0, b2);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(a1, b1);
	prod_hi = __umulhi(a1, b1);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(a2, b0);
	prod_hi = __umulhi(a2, b0);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(q0, n2);
	prod_hi = __umulhi(q0, n2);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(q1, n1);
	prod_hi = __umulhi(q1, n1);
	acc += (uint64)prod_hi << 32 | prod_lo;
	q2 = __umul24((uint32)acc, w) & LOW24;

	prod_lo = __umul24(q2, n0);
	prod_hi = __umulhi(q2, n0);
	acc = (acc + ((uint64)prod_hi << 32 | prod_lo)) >> 24;

	prod_lo = __umul24(a1, b2);
	prod_hi = __umulhi(a1, b2);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(a2, b1);
	prod_hi = __umulhi(a2, b1);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(q1, n2);
	prod_hi = __umulhi(q1, n2);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(q2, n1);
	prod_hi = __umulhi(q2, n1);
	acc += (uint64)prod_hi << 32 | prod_lo;

	r = acc & LOW24;
	acc >>= 24;

	prod_lo = __umul24(a2, b2);
	prod_hi = __umulhi(a2, b2);
	acc += (uint64)prod_hi << 32 | prod_lo;
	prod_lo = __umul24(q2, n2);
	prod_hi = __umulhi(q2, n2);
	acc += (uint64)prod_hi << 32 | prod_lo;

	prod_hi = (uint32)(acc >> 48);
	r |= acc << 24;

	if (prod_hi > 0 || r >= n)
		return r - n;
	else
		return r;

#else /* MONTMUL_RADIX == 32 */

	uint32 a0 = (uint32)a;
	uint32 a1 = (uint32)(a >> 32);
	uint32 b0 = (uint32)b;
	uint32 b1 = (uint32)(b >> 32);
	uint32 n0 = (uint32)n;
	uint32 n1 = (uint32)(n >> 32);
	uint32 acc0, acc1, acc2, nmult;
	uint32 prod_lo, prod_hi;
	uint64 prod;

	prod_lo = a0 * b0;
	prod_hi = __umulhi(a0, b0);
	acc0 = prod_lo;

	prod = (uint64)prod_hi;
	prod_lo = a1 * b0;
	prod_hi = __umulhi(a1, b0);
	prod += ((uint64)prod_hi << 32 | prod_lo);
	acc1 = (uint32)prod;
	acc2 = (uint32)(prod >> 32);

	nmult = acc0 * w;

	prod_lo = nmult * n0;
	prod_hi = __umulhi(nmult, n0);
	prod = acc0 + ((uint64)prod_hi << 32 | prod_lo);
	prod = prod >> 32;

	prod_lo = nmult * n1;
	prod_hi = __umulhi(nmult, n1);
	prod += (uint64)acc1 + ((uint64)prod_hi << 32 | prod_lo);
	acc0 = (uint32)prod;
	prod = (prod >> 32) + (uint64)acc2;
	acc1 = (uint32)prod;
	acc2 = (uint32)(prod >> 32);

	prod_lo = a0 * b1;
	prod_hi = __umulhi(a0, b1);
	prod = (uint64)acc0 + ((uint64)prod_hi << 32 | prod_lo);
	acc0 = (uint32)prod;
	prod = prod >> 32;

	prod_lo = a1 * b1;
	prod_hi = __umulhi(a1, b1);
	prod += (uint64)acc1 + ((uint64)prod_hi << 32 | prod_lo);
	acc1 = (uint32)prod;
	acc2 = (uint32)(prod >> 32) + acc2;

	nmult = acc0 * w;

	prod_hi = __umulhi(nmult, n0);
	prod_lo = nmult * n0;
	prod = acc0 + ((uint64)prod_hi << 32 | prod_lo);
	prod = prod >> 32;

	prod_hi = __umulhi(nmult, n1);
	prod_lo = nmult * n1;
	prod += acc1 + ((uint64)prod_hi << 32 | prod_lo);
	acc0 = (uint32)prod;
	prod = (prod >> 32) + (uint64)acc2;
	acc1 = (uint32)prod;
	acc2 = (uint32)(prod >> 32);

	prod = (uint64)acc1 << 32 | acc0;
	if (acc2 || prod >= n)
		return prod - n;
	else
		return prod;
#endif
}

/*------------------------------------------------------------------------*/
#if MONTMUL_RADIX == 24
	#define MONTMUL_R_LIMIT 81
#else /* MONTMUL_RADIX == 32 */
	#define MONTMUL_R_LIMIT 72
#endif

__device__ uint64 
montmul_r(uint64 n, uint32 w) {

	uint32 shift;
	uint32 i;
	uint64 shifted_n;
	uint64 res;

	shift = __clzll(n);
	shifted_n = n << shift;
	res = -shifted_n;

	for (i = 64 - shift; i < MONTMUL_R_LIMIT; i++) {
		if (res >> 63)
			res = res + res - shifted_n;
		else
			res = res + res;

		if (res >= shifted_n)
			res -= shifted_n;
	}

	res = res >> shift;
	res = montmul(res, res, n, w);
	res = montmul(res, res, n, w);
	return montmul(res, res, n, w);
}

/*------------------------------------------------------------------------*/
#define SHARED_BATCH_SIZE 48

typedef struct {
	uint32 p[SHARED_BATCH_SIZE];
	uint32 lattice_size[SHARED_BATCH_SIZE];
	uint64 roots[POLY_BATCH_SIZE][SHARED_BATCH_SIZE];
} p_soa_shared_t;

__shared__ p_soa_shared_t pbatch_cache;

__global__ void
sieve_kernel_64(p_soa_t *pbatch, 
             uint32 num_p,
	     q_soa_t *qbatch,
	     uint32 num_q,
	     uint32 num_roots,
	     found_t *found_array)
{
	uint32 my_threadid;
	uint32 num_threads;
	uint32 i, j, k, end;

	my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	num_threads = gridDim.x * blockDim.x;
	end = (num_q + num_threads - 1) / num_threads * num_threads;
	found_array[my_threadid].p = 0;

	for (i = my_threadid; i < end; i += num_threads) {
		uint32 q = i < num_q ? qbatch->p[i] : 0;
		uint64 q2 = (uint64)q * q;
		uint32 q2_w = montmul_w((uint32)q2);
		uint64 q2_r = montmul_r(q2, q2_w);
		uint32 p_done = 0;

		while (p_done < num_p) {

			uint32 curr_num_p = MIN(SHARED_BATCH_SIZE,
						num_p - p_done);

			if (threadIdx.x < curr_num_p) {
				j = threadIdx.x;

				pbatch_cache.p[j] = pbatch->p[p_done + j];
				pbatch_cache.lattice_size[j] = 
					pbatch->lattice_size[p_done + j];

				for (k = 0; k < num_roots; k++) {
					pbatch_cache.roots[k][j] = 
						pbatch->roots[k][p_done + j];
				}
			}

			__syncthreads();

			for (j = 0; j < curr_num_p && i < num_q; j++) {
				uint64 prefetch = qbatch->roots[0][i];
				uint32 p = pbatch_cache.p[j];
				uint64 p2 = (uint64)p * p;
				uint32 pinvmodq = modinv(p, q);

				uint32 lattice_size = 
						pbatch_cache.lattice_size[j];
				uint64 pinv, tmp;

				tmp = (uint64)pinvmodq * pinvmodq;
				tmp = montmul(tmp, q2_r, q2, q2_w);
				pinv = montmul(p2, tmp, q2, q2_w);
				pinv = modsub((uint64)2, pinv, q2);
				pinv = montmul(pinv, tmp, q2, q2_w);
				pinv = montmul(pinv, q2_r, q2, q2_w);

				for (k = 0; k < num_roots; k++) {

					uint64 qroot;
					uint64 proot;
					uint64 res;

					qroot = prefetch;
					prefetch = qbatch->roots[k+1][i];
					proot = pbatch_cache.roots[k][j];
					res = montmul(pinv, modsub(qroot, proot,
							q2), q2, q2_w);

					if (res < lattice_size) {
						found_t *f = found_array + 
								my_threadid;
						f->p = p;
						f->q = q;
						f->which_poly = k;
						f->offset = res;
						f->proot = proot;
					}
				}
			}

			p_done += curr_num_p;
		}
	}
}

#ifdef __cplusplus
}
#endif
