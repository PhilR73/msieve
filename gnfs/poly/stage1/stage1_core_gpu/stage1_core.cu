#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "stage1_core.h"

#ifdef __cplusplus
extern "C" {
#endif

__constant__ specialq_t q_batch[BATCH_SPECIALQ_MAX];

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_trans(uint32 *p_array, uint32 num_p, uint64 *start_roots,
			uint32 num_roots, uint32 *p_out, int64 *roots_out,
			uint32 num_specialq, uint32 num_entries)
{
	uint32 offset, i, j, p, pp_w, q, end, gcd;
	uint64 pp, pp_r, qq, tmp, inv;
	int64 newroot;

	offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (offset >= num_p)
		return;

	p = p_array[offset];
	pp = wide_sqr32(p);
	pp_w = montmul32_w((uint32)pp);
	pp_r = montmul64_r(pp, pp_w);
	end = num_p * num_roots;

	q = 0;
	for (i = 0; i < num_specialq; i++) {
		if (q != q_batch[i].p) {
			q = q_batch[i].p;
			gcd = gcd32(p, q);

			if (gcd == 1) {
				qq = wide_sqr32(q) % pp;
				tmp = modinv32(q % p, p);
				tmp = wide_sqr32(tmp);
				tmp = montmul64(tmp, pp_r, pp, pp_w);
				inv = montmul64(qq, tmp, pp, pp_w);
				inv = modsub64((uint64)2, inv, pp);
				inv = montmul64(inv, tmp, pp, pp_w);
				inv = montmul64(inv, pp_r, pp, pp_w);
			}
		}

		for (j = offset; j < end; j += num_p) {

			if (gcd == 1) {
				newroot = modsub64(start_roots[j],
						q_batch[i].root % pp, pp);
				newroot = montmul64(newroot, inv, pp, pp_w);

				if (newroot > pp / 2)
					newroot -= pp;

				p_out[j + num_entries * i] = p;
				roots_out[j + num_entries * i] = newroot;
			}
		}
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_sort(uint32 *p_array, uint64 *roots)
{
	uint32 my_threadid, offset, j, k, u, dir, tmp;
	extern __shared__ char shared_cache[];
	uint32 *p_cache;
	uint64 *root_cache;
	uint64 root_1, root_2;

	my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	offset = 2 * (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x +
							threadIdx.x;

	p_cache = (uint32 *)shared_cache;
	root_cache = (uint64 *)(p_cache + blockDim.x * 2);

	p_cache[threadIdx.x] = p_array[offset];
	p_cache[threadIdx.x + blockDim.x] = p_array[offset + blockDim.x];

	root_cache[threadIdx.x] = roots[offset];
	root_cache[threadIdx.x + blockDim.x] = roots[offset + blockDim.x];

	__syncthreads();

	for (j = 1; j <= blockDim.x; j *= 2) {

		dir = !!(my_threadid & j);
		for (k = j; k; k /= 2) {

			u = (threadIdx.x & ~(k - 1)) * 2 +
						(threadIdx.x & (k - 1));

			root_1 = root_cache[u];
			root_2 = root_cache[u + k];

			if (dir != (root_1 > root_2)) {

				tmp = p_cache[u];
				p_cache[u] = p_cache[u + k];
				p_cache[u + k] = tmp;

				root_cache[u] = root_2;
				root_cache[u + k] = root_1;
			}

			__syncthreads();
		}
	}

	p_array[offset] = p_cache[threadIdx.x];
	p_array[offset + blockDim.x] = p_cache[threadIdx.x + blockDim.x];

	roots[offset] = root_cache[threadIdx.x];
	roots[offset + blockDim.x] = root_cache[threadIdx.x + blockDim.x];
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_merge(uint32 *p_array, uint64 *roots, uint32 j)
{
	uint32 my_threadid, offset, k, u, dir, tmp;
	extern __shared__ char shared_cache[];
	uint32 *p_cache;
	uint64 *root_cache;
	uint64 root_1, root_2;

	my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	offset = 2 * (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x +
							threadIdx.x;

	p_cache = (uint32 *)shared_cache;
	root_cache = (uint64 *)(p_cache + blockDim.x * 2);

	p_cache[threadIdx.x] = p_array[offset];
	p_cache[threadIdx.x + blockDim.x] = p_array[offset + blockDim.x];

	root_cache[threadIdx.x] = roots[offset];
	root_cache[threadIdx.x + blockDim.x] = roots[offset + blockDim.x];

	__syncthreads();

	dir = !!(my_threadid & j);
	for (k = blockDim.x; k; k /= 2) {

		u = (threadIdx.x & ~(k - 1)) * 2 +
					(threadIdx.x & (k - 1));

		root_1 = root_cache[u];
		root_2 = root_cache[u + k];

		if (dir != (root_1 > root_2)) {

			tmp = p_cache[u];
			p_cache[u] = p_cache[u + k];
			p_cache[u + k] = tmp;

			root_cache[u] = root_2;
			root_cache[u + k] = root_1;
		}

		__syncthreads();
	}

	p_array[offset] = p_cache[threadIdx.x];
	p_array[offset + blockDim.x] = p_cache[threadIdx.x + blockDim.x];

	roots[offset] = root_cache[threadIdx.x];
	roots[offset + blockDim.x] = root_cache[threadIdx.x + blockDim.x];
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_merge1(uint32 *p_array, uint64 *roots, uint32 j, uint32 k)
{
	uint32 my_threadid, offset, tmp;
	uint64 root_1, root_2;

	my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	offset = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x +
							threadIdx.x;
	offset = (offset & ~(k - 1)) * 2 + (offset & (k - 1));

	root_1 = roots[offset];
	root_2 = roots[offset + k];

	if ((!!(my_threadid & j)) != (root_1 > root_2)) {

		tmp = p_array[offset];
		p_array[offset] = p_array[offset + k];
		p_array[offset + k] = tmp;

		roots[offset] = root_2;
		roots[offset + k] = root_1;
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_final(uint32 *p_array, int64 *roots, uint32 num_entries,
			uint32 num_specialq, found_t *found_array)
{
	uint32 i, my_threadid, num_threads, p_1, p_2;
	int64 root_1, root_2;

	i = my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	num_threads = gridDim.x * blockDim.x;

	while (i < num_entries * num_specialq - 1) {
		p_1 = p_array[i];
		p_2 = p_array[i + 1];
		root_1 = roots[i];
		root_2 = roots[i + 1];

		if (p_1 > 0 && p_2 > 0 && root_1 == root_2) {

			if (gcd32(p_1, p_2) == 1) {
				found_t *f = found_array + my_threadid;

				f->p1 = p_1;
				f->p2 = p_2;
				f->q = q_batch[i / num_entries].p;
				f->qroot = q_batch[i / num_entries].root;
				f->offset = root_1;
			}
		}

		i += num_threads;
	}
}

#ifdef __cplusplus
}
#endif
