#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "stage1_core_deg6_96.h"

#ifdef __cplusplus
extern "C" {
#endif

/*------------------------------------------------------------------------*/
__device__ int32
cmp96(uint96 a, uint96 b)
{
	if (a.w[2] > b.w[2])
		return 1;
	if (a.w[2] < b.w[2])
		return -1;

	if (a.w[1] > b.w[1])
		return 1;
	if (a.w[1] < b.w[1])
		return -1;

	if (a.w[0] > b.w[0])
		return 1;
	if (a.w[0] < b.w[0])
		return -1;
	return 0;
}

/*------------------------------------------------------------------------*/
__device__ uint96
add96(uint96 a, uint96 b)
{
	uint32 c;
	uint32 acc;
	uint96 res;

	acc = a.w[0] + b.w[0];
	res.w[0] = acc;
	c = (acc < a.w[0]);

	acc = a.w[1] + c;
	c = (acc < a.w[1]);
	res.w[1] = acc + b.w[1];
	c += (res.w[1] < acc);

	res.w[2] = a.w[2] + b.w[2] + c;
	return res;
}

/*------------------------------------------------------------------------*/
__device__ uint96
sub96(uint96 a, uint96 b)
{
	uint32 c;
	uint32 acc;
	uint96 res;

	acc = a.w[0] - b.w[0];
	res.w[0] = acc;
	c = (acc > a.w[0]);

	acc = a.w[1] - c;
	c = (acc > a.w[1]);
	res.w[1] = acc - b.w[1];
	c += (res.w[1] > acc);

	res.w[2] = a.w[2] - b.w[2] - c;
	return res;
}

/*------------------------------------------------------------------------*/
__device__ uint96 
modsub(uint96 a, uint96 b, uint96 p) 
{
	/* this could be 7 branch-less instructions
	   if nvcc allowed inline asm */

	uint96 res = sub96(a, b);

	if (cmp96(res, a) > 0)
		res = add96(res, p);

	return res;
}

/*------------------------------------------------------------------------*/
__device__ uint96 
wide_sqr(uint64 a)
{
	/* a < 2^48 */

	uint32 a0 = (uint32)a;
	uint32 a1 = (uint32)(a >> 32);
	uint64 acc;
	uint32 prod_lo, prod_hi;
	uint96 res;

	prod_lo = a0 * a0;
	prod_hi = __umulhi(a0, a0);
	res.w[0] = prod_lo;
	acc = (uint64)prod_hi;

	prod_lo = a0 * a1;
	prod_hi = __umulhi(a0, a1);
	acc += 2 * ((uint64)prod_hi << 32 | prod_lo);
	res.w[1] = (uint32)acc;
	res.w[2] = (uint32)(acc >> 32) + __umul24(a1, a1);

	return res;
}

/*------------------------------------------------------------------------*/
__device__ uint64 
modinv(uint64 a, uint64 p) {

	uint64 ps1, ps2, dividend, divisor, rem, q, t;
	uint32 parity;

	q = 1; rem = a; dividend = p; divisor = a;
	ps1 = 1; ps2 = 0; parity = 0;

	while (divisor > 1) {
		rem = dividend - divisor;
		t = rem - divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t; t -= divisor;
		if (rem >= divisor) { q += ps1; rem = t;
		if (rem >= divisor) {
			q = dividend / divisor;
			rem = dividend - q * divisor;
			q *= ps1;
		} } } } } } } } }

		q += ps2;
		parity = ~parity;
		dividend = divisor;
		divisor = rem;
		ps2 = ps1;
		ps1 = q;
	}
	
	if (parity == 0)
		return ps1;
	else
		return p - ps1;
}

/*------------------------------------------------------------------------*/
__device__ uint32 
montmul_w(uint32 n) {

	uint32 res = 2 + n;
	res = res * (2 + n * res);
	res = res * (2 + n * res);
	res = res * (2 + n * res);
	return res * (2 + n * res);
}

/*------------------------------------------------------------------------*/
__device__ uint96 
montmul(uint96 a, uint96 b,
		uint96 n, uint32 w) {

	uint32 acc0, acc1, acc2, acc3, nmult;
	uint32 prod_lo, prod_hi;
	uint64 prod;
	uint96 res;

	acc0 = a.w[0] * b.w[0];   /*---------------------*/
	prod = (uint64)(__umulhi(a.w[0], b.w[0]));

	prod_lo = a.w[1] * b.w[0];
	prod_hi = __umulhi(a.w[1], b.w[0]);
	prod += (uint64)prod_hi << 32 | prod_lo;
	acc1 = (uint32)prod;
	prod >>= 32;

	prod_lo = a.w[2] * b.w[0];
	prod_hi = __umulhi(a.w[2], b.w[0]);
	prod += (uint64)prod_hi << 32 | prod_lo;
	acc2 = (uint32)prod;
	acc3 = (uint32)(prod >> 32);

	nmult = acc0 * w;      /*------------------------*/

	prod_lo = nmult * n.w[0];
	prod_hi = __umulhi(nmult, n.w[0]);
	prod = ((uint64)prod_hi << 32 | prod_lo) + acc0;
	prod >>= 32;

	prod_lo = nmult * n.w[1];
	prod_hi = __umulhi(nmult, n.w[1]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc1;
	acc0 = (uint32)prod;
	prod >>= 32;

	prod_lo = nmult * n.w[2];
	prod_hi = __umulhi(nmult, n.w[2]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc2;
	acc1 = (uint32)prod;
	prod >>= 32;

	prod += acc3;
	acc2 = (uint32)prod;
	acc3 = (uint32)(prod >> 32);

	prod_lo = a.w[0] * b.w[1];   /*---------------------*/
	prod_hi = __umulhi(a.w[0], b.w[1]);
	prod = ((uint64)prod_hi << 32 | prod_lo) + acc0;
	acc0 = (uint32)prod;
	prod >>= 32;

	prod_lo = a.w[1] * b.w[1];
	prod_hi = __umulhi(a.w[1], b.w[1]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc1;
	acc1 = (uint32)prod;
	prod >>= 32;

	prod_lo = a.w[2] * b.w[1];
	prod_hi = __umulhi(a.w[2], b.w[1]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc2;
	acc2 = (uint32)prod;
	acc3 += (uint32)(prod >> 32);

	nmult = acc0 * w;      /*------------------------*/

	prod_lo = nmult * n.w[0];
	prod_hi = __umulhi(nmult, n.w[0]);
	prod = ((uint64)prod_hi << 32 | prod_lo) + acc0;
	prod >>= 32;

	prod_lo = nmult * n.w[1];
	prod_hi = __umulhi(nmult, n.w[1]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc1;
	acc0 = (uint32)prod;
	prod >>= 32;

	prod_lo = nmult * n.w[2];
	prod_hi = __umulhi(nmult, n.w[2]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc2;
	acc1 = (uint32)prod;
	prod >>= 32;

	prod += acc3;
	acc2 = (uint32)prod;
	acc3 = (uint32)(prod >> 32);

	prod_lo = a.w[0] * b.w[2];   /*---------------------*/
	prod_hi = __umulhi(a.w[0], b.w[2]);
	prod = ((uint64)prod_hi << 32 | prod_lo) + acc0;
	acc0 = (uint32)prod;
	prod >>= 32;

	prod_lo = a.w[1] * b.w[2];
	prod_hi = __umulhi(a.w[1], b.w[2]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc1;
	acc1 = (uint32)prod;
	prod >>= 32;

	prod_lo = a.w[2] * b.w[2];
	prod_hi = __umulhi(a.w[2], b.w[2]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc2;
	acc2 = (uint32)prod;
	acc3 += (uint32)(prod >> 32);

	nmult = acc0 * w;      /*------------------------*/

	prod_lo = nmult * n.w[0];
	prod_hi = __umulhi(nmult, n.w[0]);
	prod = ((uint64)prod_hi << 32 | prod_lo) + acc0;
	prod >>= 32;

	prod_lo = nmult * n.w[1];
	prod_hi = __umulhi(nmult, n.w[1]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc1;
	acc0 = (uint32)prod;
	prod >>= 32;

	prod_lo = nmult * n.w[2];
	prod_hi = __umulhi(nmult, n.w[2]);
	prod += ((uint64)prod_hi << 32 | prod_lo) + acc2;
	acc1 = (uint32)prod;
	prod >>= 32;

	prod += acc3;
	acc2 = (uint32)prod;
	acc3 = (uint32)(prod >> 32);

	res.w[0] = acc0;        /*------------------------*/
	res.w[1] = acc1;
	res.w[2] = acc2;
	if (acc3 > 0 || cmp96(res, n) >= 0)
		return sub96(res, n);
	else
		return res;
}

/*------------------------------------------------------------------------*/
__device__ uint96 
montmul_r(uint96 n, uint32 w) {

	/* 2^32 <= n < 2^96 */

	uint32 shift, word_shift, comp_shift;
	uint32 i;
	uint96 shifted_n;
	uint96 res;

	if (n.w[2] == 0) {
		shifted_n.w[2] = n.w[1];
		shifted_n.w[1] = n.w[0];
		shifted_n.w[0] = 0;
		word_shift = 32;
	}
	else {
		shifted_n = n;
		word_shift = 0;
	}

	shift = __clz(shifted_n.w[2]);
	comp_shift = 32 - shift;

	if (shift > 0) {
		shifted_n.w[2] = shifted_n.w[2] << shift | 
				shifted_n.w[1] >> comp_shift;
		shifted_n.w[1] = shifted_n.w[1] << shift | 
				shifted_n.w[0] >> comp_shift;
		shifted_n.w[0] = shifted_n.w[0] << shift;
	}

	res.w[0] = 0;
	res.w[1] = 0;
	res.w[2] = 0x80000000;
	for (i = 95 - (word_shift + shift); i < 102; i++) {
		if (res.w[2] & 0x80000000) {
			res = add96(res, res);
			res = sub96(res, shifted_n);
		}
		else {
			res = add96(res, res);
		}

		if (cmp96(res, shifted_n) > 0)
			res = sub96(res, shifted_n);
	}

	if (shift > 0) {
		res.w[0] = res.w[0] >> shift | res.w[1] << comp_shift;
		res.w[1] = res.w[1] >> shift | res.w[2] << comp_shift;
		res.w[2] = res.w[2] >> shift;
	}
	if (word_shift > 0) {
		res.w[0] = res.w[1];
		res.w[1] = res.w[2];
		res.w[2] = 0;
	}

	res = montmul(res, res, n, w);
	res = montmul(res, res, n, w);
	res = montmul(res, res, n, w);
	return montmul(res, res, n, w);
}

/*------------------------------------------------------------------------*/
__device__ p_packed_t *
p_packed_next(p_packed_t *curr)
{
	return (p_packed_t *)((uint64 *)curr + 
			P_PACKED_HEADER_WORDS + 
			3 * (curr->num_roots / 2));
}

/*------------------------------------------------------------------------*/
__constant__ uint64 pbatch[P_ARRAY_WORDS];

__constant__ uint96 two = {{2, 0, 0}};

__global__ void
sieve_kernel_96(q_soa_t *qbatch, 
             uint32 num_q,
	     uint32 num_qroots,
	     uint32 num_p,
	     found_t *found_array)
{
	uint32 my_threadid;
	uint32 num_threads;
	uint32 i, j, k, m;

	my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	num_threads = gridDim.x * blockDim.x;
	found_array[my_threadid].p = 0;

	for (i = my_threadid; i < num_q; i += num_threads) {
		uint64 q = qbatch->p[i];
		uint96 q2 = wide_sqr(q);
		uint32 q2_w = montmul_w(q2.w[0]);
		uint96 q2_r = montmul_r(q2, q2_w);
		p_packed_t *curr_p = (p_packed_t *)pbatch;
		
		for (j = 0; j < num_p; j++) {
			uint64 p = curr_p->p;
			uint96 p2 = wide_sqr(p);
			uint64 pinvmodq = modinv(p, q);

			uint32 num_proots = curr_p->num_roots;
			uint64 lattice_size = curr_p->lattice_size;
			uint96 pinv, tmp;
			uint96 test1;

			test1.w[0] = (uint32)lattice_size;
			test1.w[1] = (uint32)(lattice_size >> 32);
			test1.w[2] = 0;

			tmp = wide_sqr(pinvmodq);
			tmp = montmul(tmp, q2_r, q2, q2_w);
			pinv = montmul(p2, tmp, q2, q2_w);
			pinv = modsub(two, pinv, q2);
			pinv = montmul(pinv, tmp, q2, q2_w);
			pinv = montmul(pinv, q2_r, q2, q2_w);

			for (k = 0; k < 3 * num_qroots; k += 3) {

				uint96 qroot;

				qroot.w[0] = qbatch->roots[k][i];
				qroot.w[1] = qbatch->roots[k+1][i];
				qroot.w[2] = qbatch->roots[k+2][i];

				for (m = 0; m < num_proots; m++) {

					uint96 proot = curr_p->roots[m];
					uint96 res = montmul(pinv, 
							modsub(qroot, 
								proot, q2),
							q2, q2_w);

					if (cmp96(res, test1) <= 0) {
						found_t *f = found_array +
								my_threadid;
						f->p = p;
						f->q = q;
						f->offset = res;
						f->proot = proot;
					}
				}
			}

			curr_p = p_packed_next(curr_p);
		}
	}
}

#ifdef __cplusplus
}
#endif
